#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;

//#define _RESULT
#define _RNODE

/**
 * @ command line
 */
#define HELP_S         "-h"
#define HELP_L         "--help"
#define LOOP_S         "-l"
#define LOOP_L         "--loop"
#define PIECE_S        "-p"
#define PIECE_L        "--piece"
#define NODE_S         "-n"
#define NODE_L         "--node"
#define WIRE_S         "-w"
#define WIRE_L         "--wire"
#define PERCENT_S      "-c"
#define PERCENT_L      "-percent"
#define SEED_S         "-s"
#define SEED_L         "--seed"
#define BLOCK_S        "-b"
#define BLOCK_L        "--block"
#define THREAD_S       "-t"
#define THREAD_L       "--thread"
#define MAX_PE_S       "-m"
#define MAX_PE_L       "--maxpe"
#define PE_PER_NODE_S  "-pp"
#define PE_PER_NODE_L  "--pe-per-node"

#define PE_S           "-pe"
#define PE_L           "--proc"

/**
 * @ Operation mode
 */
#define OP_STR_WORKER  "worker"
#define OP_STR_WORKERB "workerB"
#define OP_MODE_MASTER 0x0a
#define OP_MODE_WORKER 0x0b
#define OP_MODE_WORKER_B 0x0c
int op_mode = OP_MODE_MASTER;
bool verify = false;
int parent_rank;
int wid, pe_node, rank, comm_size;
/**
 * @ data type definition
 */
#define PRECISION float
#define MPI_PRECISION MPI_FLOAT
//#define DISABLE_MATH
#define WIRE_SEGMENTS 10
#define STEPS         10000
#define DELTAT        1e-6

#define INDEX_TYPE    unsigned
#define INDEX_DIM     1

#define D_NODE        0x0000
#define D_WIRE        0x0001

int num_loops, num_pieces, nodes_per_piece;
int wires_per_piece, pct_wire_in_piece;
int random_seed, num_blocks, num_threads, num_pe, max_pe, pe_per_node;
/**
 * @ check error function
 */
inline void checkError(int ret, const char * str) {
    if (ret != 0) {
        cerr << "Error: " << str << endl;
        exit(-1);
    }
}

inline void cudaCheckError(int line, hipError_t ce)
{
    if (ce != hipSuccess){
        printf("Error: line %d %s\n", line, hipGetErrorString(ce));
        exit(1);
    }
}

/**
 * @ structure of node and wire array
 */

struct point;
typedef struct point node;
struct point {
   PRECISION * capacitance;
   PRECISION * leakage;
   PRECISION * charge;
   PRECISION * voltage; 
   int       * shr_pc;
   int       * node_attr;
};

struct edge;
typedef struct edge wire;
struct edge {
    PRECISION ** currents;
    PRECISION ** voltages;
    PRECISION *  resistance;
    PRECISION *  inductance;
    PRECISION *  capacitance;
    PRECISION *  shr_voltage;
    PRECISION *  shr_charge; 
    int       *  shr_pc;     
    int       *  in_ptr;
    int       *  out_ptr;
    int       *  wire_attr;
};

/**
 * @ Kernel Function
 */

// calculate currents gpu
__global__ void calculate_current_gpu(int num_wires, 
                PRECISION * wire_currents, PRECISION * wire_voltages, 
                int * in_ptr, int * out_ptr, 
                PRECISION * wire_inductance, PRECISION * wire_resistance, PRECISION * wire_capacitance, 
                PRECISION * node_voltage, int * wire_attr,
                PRECISION * shr_voltage) {
    int gridsize = gridDim.x * blockDim.x;
    int idx      = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_wires) {
        PRECISION temp_v[WIRE_SEGMENTS+1];
        PRECISION temp_i[WIRE_SEGMENTS];
        PRECISION old_i[WIRE_SEGMENTS];
        PRECISION old_v[WIRE_SEGMENTS-1];
        for (int it=idx; it<num_wires; it+=gridsize) {
            PRECISION dt = DELTAT;
            PRECISION recip_dt = 1.0f / dt;
            int steps = STEPS;
            int currents_offset = it * WIRE_SEGMENTS;
            int voltages_offset = it * (WIRE_SEGMENTS-1);
            // calc temporary variables
            for (int j = 0; j < WIRE_SEGMENTS; j++) {
                temp_i[j] = wire_currents[currents_offset+j];
                old_i[j]  = temp_i[j];
            }
            for (int j = 0; j < (WIRE_SEGMENTS-1); j++) {
                temp_v[j+1] = wire_voltages[voltages_offset+j];
                old_v[j]    = temp_v[j+1];
            }
            // calc outer voltages to the node voltages
            temp_v[0] = node_voltage[in_ptr[it]];
            // Note: out-ptr need communication when parallel
            if (wire_attr[it] == 0)
                temp_v[WIRE_SEGMENTS] = node_voltage[out_ptr[it]];
            else 
                temp_v[WIRE_SEGMENTS] = shr_voltage[it];
            // Solve the RLC model iteratively
            PRECISION inductance = wire_inductance[it];
            PRECISION recip_resistance = 1.0f / (wire_resistance[it]);
            PRECISION recip_capacitance = 1.0f / (wire_capacitance[it]);
            for (int j = 0; j < steps; j++) {
                // first, figure out the new current from the voltage differential
                // and our inductance:
                // dV = R*I + L*I' ==> I = (dV - L*I')/R
                for (int k = 0; k < WIRE_SEGMENTS; k++) {
                    temp_i[k] = ((temp_v[k+1] - temp_v[k]) - (inductance * (temp_i[k] - old_i[k]) * recip_dt)) * recip_resistance;
                }
                // Now update the inter-node voltages
                for (int k = 0; k < (WIRE_SEGMENTS-1); k++) {
                    temp_v[k+1] = old_v[k] + dt * (temp_i[k] - temp_i[k+1]) * recip_capacitance;
                }
            }
            // Write out the results
            for (int j = 0; j < WIRE_SEGMENTS; j++)
                wire_currents[currents_offset+j] = temp_i[j];
            for (int j = 0; j < (WIRE_SEGMENTS-1); j++)
                wire_voltages[voltages_offset+j] = temp_v[j+1];
        }// for: wires
    }// if
    __syncthreads();
}// calc_end

// distributed charge gpu
__global__ void distributed_charge_gpu(int num_wires, 
                PRECISION * wire_currents,  
                int * in_ptr, int * out_ptr, 
                PRECISION * node_charge, int * wire_attr,
                PRECISION * shr_charge) {
    int gridsize = gridDim.x * blockDim.x;
    int idx      = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_wires) {
        for (int it = idx; it < num_wires; it+=gridsize) {
            int currents_offset = it * WIRE_SEGMENTS;
            // calc temporary variables
            PRECISION dt = DELTAT;
            PRECISION in_current = -dt * (wire_currents[currents_offset]);
            PRECISION out_current = -dt * (wire_currents[currents_offset+WIRE_SEGMENTS-1]);
            //node_charge[in_ptr[it]]  += in_current;
            atomicAdd(&node_charge[in_ptr[it]], in_current);
            //node_charge[out_ptr[it]] += out_current;
            if (wire_attr[it] == 0)
                atomicAdd(&node_charge[out_ptr[it]], out_current);
            else
                atomicAdd(&shr_charge[it], out_current);
        }//for: iterate wires_per_piece
    }// if
    __syncthreads();
}// dc end
// update voltage gpu
__global__ void update_voltage_gpu( int num_nodes,
                PRECISION * node_voltage, PRECISION * node_charge, 
                PRECISION * node_capacitance, PRECISION * node_leakage) {
    int gridsize = gridDim.x * blockDim.x;
    int idx      = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_nodes) {
        for (int it = idx; it < num_nodes; it+=gridsize) {
            PRECISION voltage = node_voltage[it];
            PRECISION charge = node_charge[it];
            PRECISION capacitance = node_capacitance[it];
            PRECISION leakage = node_leakage[it];
            voltage += charge / capacitance;
            voltage *= (1.f - leakage);
            //node_pc[n].voltage[it] = voltage;
            node_voltage[it] = voltage;
            node_charge[it]  = 0.f;
        }//for: iterate nodess_per_piece
    }//if
    __syncthreads();
}

/**
 * @ random function to get node
 */

int random_element(int vec_size)
{
  int index = int(drand48() * vec_size);
  return index;
}

void node_init(node input, int ioffset, node * output, int ooffset, int param_size) {
    for (int i=0; i<param_size; i++) {
        output[ooffset].capacitance = input.capacitance + i+ioffset;
        output[ooffset].leakage     = input.leakage     + i+ioffset;
        output[ooffset].charge      = input.charge      + i+ioffset;
        output[ooffset].voltage     = input.voltage     + i+ioffset;
        output[ooffset].node_attr   = input.node_attr   + i+ioffset;
    }
}
void wire_init(wire input, int ioffset, wire * output, int ooffset, int param_size) {
    for (int i=0; i<param_size; i++) {
        output[ooffset].currents    = input.currents + i+ioffset;
        output[ooffset].voltages    = input.voltages + i+ioffset;
        output[ooffset].resistance  = input.resistance + i+ioffset;
        output[ooffset].inductance  = input.inductance+ i+ioffset;
        output[ooffset].capacitance = input.capacitance + i+ioffset;
        output[ooffset].in_ptr      = input.in_ptr   + i+ioffset;
        output[ooffset].out_ptr     = input.out_ptr  + i+ioffset;
        output[ooffset].shr_pc      = input.shr_pc   + i+ioffset;
        output[ooffset].shr_voltage = input.shr_voltage + i+ioffset;
        output[ooffset].shr_charge  = input.shr_charge  + i+ioffset;
        output[ooffset].wire_attr   = input.wire_attr   + i+ioffset;

    }
}

void getConfig(int argc, char ** argv){

    if (argc == 1){
        cout << "\n==== HELP ====\n-h or --help\tfor help\n-l or --loop\tto set loop times\n"
            "-p or --pieces\tto set pieces\n-n or --nodes\tto specify number of nodes\n"
            "-w or --wires\tto specify number of wires\n"
            "-c or --percent\tto specify pencentage of private nodes\n"
            "-s or --seed\tto specify random seed\n"
            "-b or --block\tto specify number of block\n"
            "-pe or --proc\tto specify number of process\n"
            "-m or --maxpe\tto specify max number of process\n"
            "-pp or --pe-per-node\tto specify number of process per node\n"
            "-t or --thread\tto speicify number of thread\n\n";
        exit(-1);
    }

    for (int i = 1; i < argc; i++){
        if ( !strcmp(argv[i], HELP_S) || !strcmp(argv[i], HELP_L) ) {

            cout << "\n==== HELP ====\n-h or --help\tfor help\n-l or --loop\tto set loop times\n"
                "-p or --pieces\tto set pieces\n-n or --nodes\tto specify number of nodes\n"
                "-w or --wires\tto specify number of wires\n"
                "-c or --percent\tto specify pencentage of private nodes\n"
                "-s or --seed\tto specify random seed\n"
                "-b or --block\tto specify number of block\n"
                "-pe or --proc\tto specify number of process\n"
                "-m or --maxpe\tto specify max number of process\n"
                "-pp or --pe-per-node\tto specify number of process per node\n"
                "-t or --thread\tto speicify number of thread\n\n";
            exit(-1);
        }
        else if ( !strcmp(argv[i], LOOP_S) || !strcmp(argv[i], LOOP_L) ) {
            num_loops = atoi(argv[i + 1]);
            i++;
        }
        else if ( !strcmp(argv[i], PIECE_S) || !strcmp(argv[i], PIECE_L) ) {
            num_pieces = atoi(argv[i + 1]);
            i++;
        }
        else if ( !strcmp(argv[i], NODE_S) || !strcmp(argv[i], NODE_L) ) {
            nodes_per_piece = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], WIRE_S) || !strcmp(argv[i], WIRE_L)){
            wires_per_piece = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], PERCENT_S) || !strcmp(argv[i], PERCENT_L)){
            pct_wire_in_piece = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], SEED_S) || !strcmp(argv[i], SEED_L)){
            random_seed = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], BLOCK_S) || !strcmp(argv[i], BLOCK_L)){
            num_blocks = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], THREAD_S) || !strcmp(argv[i], THREAD_L)){
            num_threads = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], PE_S) || !strcmp(argv[i], PE_L)){
            num_pe = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], MAX_PE_S) || !strcmp(argv[i], MAX_PE_L)){
            max_pe = atoi(argv[i + 1]);
            i++;
        }
        else if (!strcmp(argv[i], PE_PER_NODE_S) || !strcmp(argv[i], PE_PER_NODE_L)){
            pe_per_node = atoi(argv[i + 1]);
            i++;
        }
        else {
            cout << "Unknow parameter!" << endl;
            exit(-1);
        }
    }
}

int SetOperationMode(int argc, char ** argv) {
    bool matched   = false;
    for (int idx=0;idx<argc;idx++) {
        if (!strcmp(argv[idx], OP_STR_WORKER)) {
            op_mode     = OP_MODE_WORKER;
            matched     = true;
            parent_rank = atoi(argv[idx + 1]);
            wid         = atoi(argv[idx + 2]);
        }
        if (!strcmp(argv[idx], OP_STR_WORKERB)) {
            op_mode     = OP_MODE_WORKER_B;
            matched     = true;
            parent_rank = atoi(argv[idx + 1]);
            wid         = atoi(argv[idx + 2]);
        }
        if (!strcmp(argv[idx], "-m") && op_mode == OP_MODE_MASTER) {
            max_pe  = atoi(argv[idx+1]);
        }
        if (!strcmp(argv[idx], "-pn") && op_mode == OP_MODE_MASTER) {
            pe_node = atoi(argv[idx+1]);
        }
        if (!strcmp(argv[idx], "-pp") && op_mode == OP_MODE_MASTER) {
            pe_per_node = atoi(argv[idx+1]);
        }
        if (!strcmp(argv[idx], "-v") && op_mode == OP_MODE_MASTER) {
           verify   = true;
        }
    }
    if (!matched)
        op_mode = OP_MODE_MASTER;
}

int MasterRoutine(int argc, char ** argv);
int WorkerRoutine(int argc, char ** argv);
int WorkerRoutine_B(int argc, char ** argv);

int main(int argc, char ** argv) {
  
/* parameter setting */
    num_loops         = 1;
    num_pieces        = 4;
    nodes_per_piece   = 2;
    wires_per_piece   = 4;
    pct_wire_in_piece = 95;
    random_seed       = 0;
    num_blocks        = 32;
    num_threads       = 256;
    
    //getConfig(argc, argv);

/* MPI init */
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_size); 
    SetOperationMode(argc, argv); 

    num_pe = comm_size;
    int ret;
    switch(op_mode) {
    case OP_MODE_MASTER:
        ret = MasterRoutine(argc, argv);
        break;
    case OP_MODE_WORKER:
        ret = WorkerRoutine(argc, argv);
        break;
    case OP_MODE_WORKER_B:
        ret = WorkerRoutine_B(argc, argv);
        break;
    }

    return 0;
}

int MasterRoutine(int argc, char ** argv) {
    getConfig(argc, argv);

    MPI_Barrier(MPI_COMM_WORLD);

    // set random seed
    srand48(random_seed);

//    int random_seed       = 12345;
    int steps             = STEPS;

    long num_circuit_nodes = num_pieces * nodes_per_piece;
    long num_circuit_wires = num_pieces * wires_per_piece;

    // calculate currents
    long operations = num_circuit_wires * (WIRE_SEGMENTS*6 + (WIRE_SEGMENTS-1)*4) * steps;
    // distribute charge
    operations += (num_circuit_wires * 4);
    // update voltages
    operations += (num_circuit_nodes * 4);
    // multiply by the number of loops
    operations *= num_loops;
    // Control PE master PE rank = 0
    int count = 0;
    int count_buf[3];
    int child_buf[2];
    int control_buf[4];
    int compute_info[3];
    bool end = false;
    int rev_id = 0;

    char rank_str[8], wid_str[8];
    snprintf(rank_str, 8, "%d", rank);
    MPI_Request request;
    MPI_Status  status;
    memset(count_buf, 0, sizeof(count_buf));
    memset(child_buf, 0, sizeof(child_buf));
    memset(control_buf, 0, sizeof(control_buf));
    memset(compute_info, 0, sizeof(compute_info));


/* worksplit */
    int pieces_per_pe = num_pieces / max_pe;
/* circuit graph building */
    // node definition 
    node * node_piece;
    node * first_nodes;
    wire * wire_piece;
    wire * first_wires;
    if (rank == 0) {
        node_piece  = new node[num_pieces];
        first_nodes = new node[num_pieces];
        vector<int> shared_nodes_piece(num_pieces, 0); 
        // wire definition
        wire_piece = new wire[num_pieces];
        first_wires = new wire[num_pieces];
        // node initialization
        for (int n = 0; n < num_pieces; n++) {
            // allocate space for array in soa
            node_piece[n].capacitance = new PRECISION[nodes_per_piece];
            node_piece[n].leakage     = new PRECISION[nodes_per_piece];
            node_piece[n].charge      = new PRECISION[nodes_per_piece];
            node_piece[n].voltage     = new PRECISION[nodes_per_piece];
            node_piece[n].shr_pc      = new int[nodes_per_piece];
            node_piece[n].node_attr   = new int[nodes_per_piece];
            // initialize node_per_piece
            for (int i = 0; i < nodes_per_piece; i++) {
                // initialize node parameter
                node_piece[n].capacitance[i] = drand48() + 1.f;
                node_piece[n].leakage[i]     = 0.1f * drand48();
                node_piece[n].charge[i]      = 0.f;
                node_piece[n].voltage[i]     = 2*drand48() - 1.f;
                // node_attr (0:private, 1:shared)
                node_piece[n].shr_pc[i]      = 0;
                node_piece[n].node_attr[i]   = 0;
            
                // set first node in each piece
                if (i == 0) {
                    // allocate space for first node
                    // initialize first node
                    node_init(node_piece[n], i, first_nodes, n, 1);
                } //if
            }//for
        }//for

        // wire initialization
        for (int n = 0; n < num_pieces; n++) {
#ifdef _RESULT
            printf("=== List nodes in piece %d ===\n", n);
#endif
            // allocate space for array in soa of wire
            wire_piece[n].currents    = new PRECISION*[wires_per_piece];
            for (int j=0; j<wires_per_piece; j++)
                wire_piece[n].currents[j]  = new PRECISION[WIRE_SEGMENTS];
            wire_piece[n].voltages    = new PRECISION*[wires_per_piece];
            for (int j=0; j<wires_per_piece; j++)
                wire_piece[n].voltages[j]  = new PRECISION[WIRE_SEGMENTS-1];
            wire_piece[n].resistance  = new PRECISION[wires_per_piece];
            wire_piece[n].inductance  = new PRECISION[wires_per_piece];
            wire_piece[n].capacitance = new PRECISION[wires_per_piece];
            wire_piece[n].in_ptr      = new int[wires_per_piece];
            wire_piece[n].out_ptr     = new int[wires_per_piece];
            wire_piece[n].wire_attr   = new int[wires_per_piece];
            // init wire shared part
            wire_piece[n].shr_voltage = new PRECISION[wires_per_piece];
            wire_piece[n].shr_charge  = new PRECISION[wires_per_piece];
            wire_piece[n].shr_pc      = new int[wires_per_piece];
            for (int j=0; j<wires_per_piece; j++) {
                wire_piece[n].shr_voltage[j] = 0.f;
                wire_piece[n].shr_charge[j]  = 0.f;
                wire_piece[n].shr_pc[j]      = 0;
                wire_piece[n].wire_attr[j]   = 0;
            }
            // initialize wire parameter
            for (int i = 0; i < wires_per_piece; i++) {
                // init currents
                for (int j = 0; j < WIRE_SEGMENTS; j++)
                    wire_piece[n].currents[i][j] = 0.f;
                // init voltage
                for (int j = 0; j < WIRE_SEGMENTS-1; j++)
                    wire_piece[n].voltages[i][j] = 0.f;
                // init resistance
                wire_piece[n].resistance[i]  = drand48() * 10.0 + 1.0;
                // Keep inductance on the order of 1e-3 * dt to avoid resonance problems
                wire_piece[n].inductance[i]  = (drand48() + 0.1) * DELTAT * 1e-3;
                wire_piece[n].capacitance[i] = drand48() * 0.1;
                // UNC init connection
                wire_piece[n].in_ptr[i] = random_element(nodes_per_piece);
                //node_init(node_piece[n], random_element(nodes_per_piece), wire_piece[n].in_ptr, i, 1);
//            wire_piece[n].in_ptr[i][0] = random_element(nodes_per_piece);
                if ((100 * drand48()) < pct_wire_in_piece) {
                    wire_piece[n].out_ptr[i] = random_element(nodes_per_piece);
                //node_init(node_piece[n], random_element(nodes_per_piece), wire_piece[n].out_ptr, i, 1);
//                wire_piece[n].back().out_ptr = random_element(nodes_per_piece);
                }//if
                else {
#ifdef _RESULT
                    cout << "\t\tShared appear\n";
#endif
                    // make wire as shared
                    wire_piece[n].wire_attr[i] = 1;
                    //node_piece[n].node_attr[wire_piece[n].in_ptr[i]] = 1;
                    //*((wire_piece[n].in_ptr[i]).node_attr) = 1;
                    // pick a random other piece and a node from there
                    int nn = int(drand48() * (num_pieces - 1));
                    if (nn >= n) nn++;
                    // pick an arbitrary node, except that if it's one that didn't used to be shared, make the 
                    //  sequentially next pointer shared instead so that each node's shared pointers stay compact
                    int idx = int(drand48() * nodes_per_piece);
                    if (idx > shared_nodes_piece[nn])
                        idx = shared_nodes_piece[nn]++;
                    // mark idx node of this piece the shr piece info 
                    wire_piece[n].shr_pc[i] = nn;
                    // make output node as shared and record shared peieces
                    node_piece[nn].shr_pc[idx]    = n;
                    node_piece[nn].node_attr[idx] = 1;
                    
                    wire_piece[n].out_ptr[i] = idx;
                }//else
                // Record the first wire pointer for this piece
                if (i == 0)
                    wire_init(wire_piece[n], i, first_wires, n, 1);
#ifdef _RESULT
                // circuit info
                printf( "Wire %d resistance: %f, inductance: %f, capacitance: %f\n", i, wire_piece[n].resistance[i], wire_piece[n].inductance[i], wire_piece[n].capacitance[i]);
                printf("** node info **\n");
                printf("in_ptr/node_type:%d, capacitance: %f\n", node_piece[n].node_attr[(wire_piece[n].in_ptr[i])], node_piece[n].capacitance[(wire_piece[n].in_ptr[i])]);
                printf("out_ptr/node_type:%d, capacitance: %f\n", node_piece[n].node_attr[(wire_piece[n].out_ptr[i])], node_piece[n].capacitance[(wire_piece[n].out_ptr[i])]);
#endif
            }//for: wire_per_piece
        }//for : pieces
    }
    else {
        int pieces_per_work = pieces_per_pe * pe_per_node;
        node_piece  = new node[pieces_per_work];
        first_nodes = new node[pieces_per_work];
        // wire definition
        wire_piece = new wire[pieces_per_work];
        first_wires = new wire[pieces_per_work];
        // node initialization
        for (int n = 0; n < pieces_per_work; n++) {
            // allocate space for array in soa
            node_piece[n].capacitance = new PRECISION[nodes_per_piece];
            node_piece[n].leakage     = new PRECISION[nodes_per_piece];
            node_piece[n].charge      = new PRECISION[nodes_per_piece];
            node_piece[n].voltage     = new PRECISION[nodes_per_piece];
            node_piece[n].shr_pc      = new int[nodes_per_piece];
            node_piece[n].node_attr   = new int[nodes_per_piece];
        }
        // wire initialization
        for (int n = 0; n < pieces_per_work; n++) {
            // allocate space for array in soa of wire
            wire_piece[n].currents    = new PRECISION*[wires_per_piece];
            for (int j=0; j<wires_per_piece; j++)
                wire_piece[n].currents[j]  = new PRECISION[WIRE_SEGMENTS];
            wire_piece[n].voltages    = new PRECISION*[wires_per_piece];
            for (int j=0; j<wires_per_piece; j++)
                wire_piece[n].voltages[j]  = new PRECISION[WIRE_SEGMENTS-1];
            wire_piece[n].resistance  = new PRECISION[wires_per_piece];
            wire_piece[n].inductance  = new PRECISION[wires_per_piece];
            wire_piece[n].capacitance = new PRECISION[wires_per_piece];
            wire_piece[n].in_ptr      = new int[wires_per_piece];
            wire_piece[n].out_ptr     = new int[wires_per_piece];
            wire_piece[n].wire_attr   = new int[wires_per_piece];
            // init wire shared part
            wire_piece[n].shr_voltage = new PRECISION[wires_per_piece];
            wire_piece[n].shr_charge  = new PRECISION[wires_per_piece];
            wire_piece[n].shr_pc      = new int[wires_per_piece];
        }
    }
    // global synchronization
    MPI_Barrier(MPI_COMM_WORLD);
    /* GPU main loop */
    // control parameter settings
    char hostname[256];
    gethostname(hostname, 256);

for (int iloop = 0; iloop < num_loops; iloop++) {

    /*** Dynamic scheduling procedure  ***/
    // control parameter settings
    count = 0;
    /* ditribute data to PEs */
    if (rank == 0) {
#ifdef _RNODE
            printf("My rank: %d, Node name: %s\n", rank, hostname); 
#endif
        //Note: put this for loop outside 
        //for (int iloop = 0; iloop < num_loops; iloop++) { 
            memset(count_buf, 0, sizeof(count_buf));
            memset(child_buf, 0, sizeof(child_buf));
            memset(control_buf, 0, sizeof(control_buf));
            memset(compute_info, 0, sizeof(compute_info));
            while (count < max_pe) {
                MPI_Recv(control_buf, 4, MPI_INT, MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, &status);
                
                rev_id = status.MPI_SOURCE;
                if (rev_id == -1) {
                    cout << "Error: Master PE Monitor Receiving Error!" << endl;
                    exit(0);
                }
                // work not finish yet or beginning
                if (control_buf[0] == 1) {
                    // count_buf[1] new request
                    count_buf[1] = 1;
                    // count_buf[2] WID for new child PE
                    count_buf[2] = count;
                    // response new child PE and send WID                   
                    MPI_Send(count_buf, 3, MPI_INT, rev_id, 0, MPI_COMM_WORLD);
                    // contorl_buf[2] result ready
                    if (control_buf[2] == 1) {
                        //post_work
                        for (int p=0; p<pe_per_node; p++) {
                            int pwid = control_buf[1] + p;
                            if (pwid < max_pe) {
                                for (int pwc=0; pwc<pieces_per_pe; pwc++) {
                                    int poffset = pwid*pieces_per_pe + pwc;
                                    for (int i=0; i<wires_per_piece; i++) 
                                        MPI_Recv(wire_piece[poffset].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Recv(wire_piece[poffset].voltages[i], WIRE_SEGMENTS-1, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    MPI_Recv(node_piece[poffset].charge     , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    MPI_Recv(wire_piece[poffset].shr_charge , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                }//endfor pwc 
                            }//endif max_pe
                        }//endfor
                    }//endif
                    // identify work portition to new child PE
                    for (int p=0; p<pe_per_node; p++) {
                        wid = count + p;
                        if (wid < max_pe) {
                            for (int c=0; c<pieces_per_pe; c++) {
                                int noffset = wid*pieces_per_pe + c;
                                // send node info
                                MPI_Send(node_piece[noffset].capacitance, nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].leakage    , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].voltage    , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].charge     , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].shr_pc     , nodes_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].node_attr  , nodes_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                // send wire info
                                for (int i=0; i<wires_per_piece; i++)
                                    MPI_Send(wire_piece[noffset].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                for (int i=0; i<wires_per_piece; i++)
                                    MPI_Send(wire_piece[noffset].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].resistance , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].inductance , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].capacitance, wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].in_ptr     , wires_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].out_ptr    , wires_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].wire_attr  , wires_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].shr_pc     , wires_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].shr_voltage, wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].shr_charge , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                            }//endfor pieces_per_pe
                        }//endif max_pe
                    }//endfor pe_per_node
                    count += pe_per_node;
                    control_buf[0] = 0 ;
                    control_buf[1] = 0;
                    control_buf[2] = 0;
                }//endif work finish
            }//endwhile
            // get final result 
            for (int i=1; i < num_pe; i++) {
                MPI_Recv(control_buf, 4, MPI_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                count_buf[1] = 2;
                MPI_Send(count_buf, 3, MPI_INT, i, 0, MPI_COMM_WORLD);
                int inum = i;
                // post_work
                for (int p=0; p<pe_per_node; p++) {
                    int pwid = control_buf[1] + p;
                    if (pwid < max_pe) {
                        for (int pwc=0; pwc<pieces_per_pe; pwc++) {
                            int poffset = pwid*pieces_per_pe + pwc;
                            for (int i=0; i<wires_per_piece; i++)
                                MPI_Recv(wire_piece[poffset].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, inum, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                            for (int i=0; i<wires_per_piece; i++)
                                MPI_Recv(wire_piece[poffset].voltages[i], WIRE_SEGMENTS-1, MPI_PRECISION, inum, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                            MPI_Recv(node_piece[poffset].charge     , nodes_per_piece, MPI_PRECISION, inum, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                            MPI_Recv(wire_piece[poffset].shr_charge , wires_per_piece, MPI_PRECISION, inum, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                        }//endfor pwc 
                    }//endif max_pe
                }//endfor

            } 
            // Note: done by single main PE only, post work to update node charge 
#if 1
            for (int n = 0; n < num_pieces; n++) {
                for (int i=0; i<wires_per_piece; i++) {
                    if (wire_piece[n].wire_attr[i] == 1) {
                        node_piece[wire_piece[n].shr_pc[i]].charge[wire_piece[n].out_ptr[i]] += wire_piece[n].shr_charge[i];
                        wire_piece[n].shr_charge[i] = 0.f;
                    }// endif wire_attr
                }// endfor n_num_piece
            } //end for num_pieces
#endif
#if 0
            for (int n=0; n<num_pieces; n++) {
                for (int it=0; it<nodes_per_piece; it++) {
                    printf("\t**node info **\n");
                    printf("\tvoltage: %f, charge: %f\n", node_piece[n].voltage[it], node_piece[n].charge[it]);
                }
            }
#endif
    }//endif rank0
    else {
#ifdef _RNODE
        printf("My rank: %d, Node name: %s\n", rank, hostname); 
#endif
        int  pieces_per_work = pe_per_node * pieces_per_pe;
            memset(count_buf, 0, sizeof(count_buf));
            memset(child_buf, 0, sizeof(child_buf));
            memset(control_buf, 0, sizeof(control_buf));
            memset(compute_info, 0, sizeof(compute_info));
            end = false;
            control_buf[3] = 1;
            while (!end) {
                // request new child PE
                control_buf[0] = 1;
                MPI_Send(control_buf, 4, MPI_INT, 0, 0, MPI_COMM_WORLD);
                // receive child PE request, wid or complete
                MPI_Recv(count_buf, 3, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                // get wid, begin computation
                if (count_buf[1] == 1) {
                    if (control_buf[2] == 1) {
                        //post_work
                        for (int p=0; p<pe_per_node; p++) {
                            int pwid = wid + p;
                            if (pwid < max_pe) {
                                for (int n = 0; n < pieces_per_pe; n++) {
                                    int npiece = p*pieces_per_pe + n;
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                }// end for
                            }//endif
                        }// end for
                    }//endif
                    // init wid
                    wid = count_buf[2];
                    
                    for (int p=0; p<pe_per_node; p++) {
                        int wtid = wid + p;
                        if (wtid < max_pe) {         
                            for (int n = 0; n < pieces_per_pe; n++) {
                                int npiece = p*pieces_per_pe + n;
                                // allocate space for array in soa
                               MPI_Recv(node_piece[npiece].capacitance, nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(node_piece[npiece].leakage    , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(node_piece[npiece].voltage    , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(node_piece[npiece].shr_pc     , nodes_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(node_piece[npiece].node_attr  , nodes_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               for (int i=0; i<wires_per_piece; i++)
                                   MPI_Recv(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               for (int i=0; i<wires_per_piece; i++)
                                   MPI_Recv(wire_piece[npiece].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].resistance , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].inductance , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].capacitance, wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].in_ptr     , wires_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].out_ptr    , wires_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].wire_attr  , wires_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].shr_pc     , wires_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].shr_voltage, wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                            }//endfor pieces_per_pe
                        }//endif avail pe
                    }//endfor pe_per_node
                    
                    MPI_Comm   children_comm;
                    // spawn child pe to run
                    snprintf(wid_str, 8, "%d", count_buf[2]);
                    char     * c_argv[] = {const_cast<char *>("worker"), rank_str, wid_str, NULL};
                    //Each of the processes in the master-job spawns a worker-job
                    //consisting of NUM_WORKER_PROCS processes.
                    // spawn info
#if 0
                    int offset=0;
                    for (int i=0;i<strlen(hostname);i++) {
                        if (hostname[i] == '.') {
                            offset = i;
                            break;
                        }
                    }
                    hostname[offset] = '\0';
#endif
                    MPI_Info spawn_info;
                    MPI_Info_create(&spawn_info);
                    MPI_Info_set(spawn_info, "host", hostname);

                    int pe_spn;
                    if (wid+pe_per_node > max_pe)
                        pe_spn = max_pe - wid;
                    else
                        pe_spn = pe_per_node;

#if 0
                    printf("PE[%d], NODE[%s], mPE pe_spn: %d, pieces_per_pe: %d, nodes_per_piece: %d\n", rank, hostname, pe_spn, pieces_per_pe, nodes_per_piece);
#endif
                    //DEBUG_R
                    MPI_Comm_spawn(argv[0], c_argv, pe_spn, spawn_info, 0, MPI_COMM_SELF, &children_comm, MPI_ERRCODES_IGNORE);
                    //MPI_Comm_spawn(argv[0], c_argv, pe_spn, MPI_INFO_NULL, 0, MPI_COMM_SELF, &children_comm, MPI_ERRCODES_IGNORE);
                    //send computation info to children PEs
                    compute_info[0] = pieces_per_pe;
                    compute_info[1] = nodes_per_piece;
                    compute_info[2] = wires_per_piece;
#ifdef DEBUGER_OUT
                    printf("\tPE[%d], NODE[%s], currwid[%d]\n", rank, hostname, wid);
#endif
                    // send data
                    for (int p=0; p<pe_spn; p++) {
                        int wtid = wid + p;
                        //if (wtid < max_pe) {
#if 1
                            MPI_Send(compute_info, 3, MPI_INT, p, 0, children_comm);
#else
                            MPI_Bcast(compute_info, 2, MPI_INT, MPI_ROOT, children_comm);
#endif
                            for (int n = 0; n < pieces_per_pe; n++) {
                                int npiece = p*pieces_per_pe + n;
                                // allocate space for array in soa
#if 1
                                MPI_Send(node_piece[npiece].capacitance, nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].leakage    , nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].voltage    , nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].shr_pc     , nodes_per_piece, MPI_INT      , p, 0, children_comm);
                                MPI_Send(node_piece[npiece].node_attr  , nodes_per_piece, MPI_INT      , p, 0, children_comm);
#if 1
                               for (int i=0; i<wires_per_piece; i++)
                                   MPI_Send(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, p, 0, children_comm);
                               for (int i=0; i<wires_per_piece; i++)
                                   MPI_Send(wire_piece[npiece].voltages[i], WIRE_SEGMENTS-1, MPI_PRECISION, p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].resistance , wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].inductance , wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].capacitance, wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].in_ptr     , wires_per_piece, MPI_INT      , p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].out_ptr    , wires_per_piece, MPI_INT      , p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].wire_attr  , wires_per_piece, MPI_INT      , p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].shr_voltage, wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                               MPI_Send(wire_piece[npiece].shr_pc     , wires_per_piece, MPI_INT      , p, 0, children_comm);
#endif
#else
#if 0
                                MPI_Bcast(node_piece[npiece].capacitance, nodes_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                                MPI_Bcast(node_piece[npiece].leakage    , nodes_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                                MPI_Bcast(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                                MPI_Bcast(node_piece[npiece].voltage    , nodes_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                                MPI_Bcast(node_piece[npiece].shr_pc     , nodes_per_piece, MPI_INT      , MPI_ROOT, children_comm);
                                MPI_Bcast(node_piece[npiece].node_attr  , nodes_per_piece, MPI_INT      , MPI_ROOT, children_comm);
                               for (int i=0; i<wires_per_piece; i++)
                                   MPI_Bcast(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, MPI_ROOT, children_comm);
                               for (int i=0; i<wires_per_piece; i++)
                                   MPI_Bcast(wire_piece[npiece].voltages[i], WIRE_SEGMENTS-1, MPI_PRECISION, MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].resistance , wires_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].inductance , wires_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].capacitance, wires_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].in_ptr     , wires_per_piece, MPI_INT      , MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].out_ptr    , wires_per_piece, MPI_INT      , MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].wire_attr  , wires_per_piece, MPI_INT      , MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].shr_voltage, wires_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
                               MPI_Bcast(wire_piece[npiece].shr_pc     , wires_per_piece, MPI_PRECISION, MPI_ROOT, children_comm);
#endif
#endif
                            }//endfor pieces_per_pe
                        //}//endif avail pe
                    }//endfor pe_per_node
                    // receive data back
                    for (int p=0; p<pe_per_node; p++) {
                        int wtid = wid + p;
                        if (wtid < max_pe) {
                            // get post_work back
                            for (int n=0; n<pieces_per_pe; n++) {
                                int npiece = p*pieces_per_pe + n;
                                for (int i=0; i<wires_per_piece; i++)
                                    MPI_Recv(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                for (int i=0; i<wires_per_piece; i++)
                                    MPI_Recv(wire_piece[npiece].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[npiece].charge, nodes_per_piece, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                MPI_Recv(wire_piece[npiece].shr_charge, wires_per_piece, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                            } //end for num_piece
                        } //endif wtid
                    } // end for pe_per_node
                    //Disconnect children PE
                    MPI_Comm_disconnect(&children_comm);
#ifdef DEBUGER_OUT
                    printf("\t Rank: %d, Exit working PE.\n", rank);
#endif
                    // set count_buf[0] to transfer results back to main PE
                    count_buf[0] = 1; 
                    control_buf[1] = wid;
                    control_buf[2] = 1;
                }//endif computation
                else if (count_buf[1] == 2) {
#ifdef DEBUGER_OUT
                    printf("\t Rank: %d, Enter working PE final...\n", rank);
#endif
                    if (control_buf[2] == 1) {
#ifdef DEBUGER_OUT
                        printf("\t Rank: %d, Enter working PE commit...\n", rank);
#endif
                        for (int p=0; p<pe_per_node; p++) {
                            int wtid = wid + p;
                            if (wtid < max_pe) {
                               for (int n = 0; n < pieces_per_pe; n++) {
                                    int npiece = p*pieces_per_pe + n;
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                               }// end for
                            }//endif
                        }//endfor 
#ifdef DEBUGER_OUT
                        printf("\t Rank: %d, Exit working PE commit.\n", rank);
#endif
                    }//endif send final work back
                    control_buf[2] = 0;
                    end = true;
                }//endelseif endflag communication
            }//end while
    }//endelse
    MPI_Barrier(MPI_COMM_WORLD);
#ifdef _RNODE
    printf("End of first phase computing---> PE:%d\n", rank);
#endif
/***** The second-phase computation *****/
    // Note: second-phase work 
#if 1
    /*** Dynamic scheduling procedure  ***/
    // control parameter settings
    count = 0;
    /* ditribute data to PEs */
    if (rank == 0) {
            memset(count_buf, 0, sizeof(count_buf));
            memset(child_buf, 0, sizeof(child_buf));
            memset(control_buf, 0, sizeof(control_buf));
            memset(compute_info, 0, sizeof(compute_info));
            while (count < max_pe) {
                MPI_Recv(control_buf, 4, MPI_INT, MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, &status);
                
                rev_id = status.MPI_SOURCE;
                if (rev_id == -1) {
                    cout << "Error: Master PE Monitor Receiving Error!" << endl;
                    exit(0);
                }
                // work not finish yet or beginning
                if (control_buf[0] == 1) {
                    // count_buf[1] new request
                    count_buf[1] = 1;
                    // count_buf[2] WID for new child PE
                    count_buf[2] = count;
                    // response new child PE and send WID                   
                    MPI_Send(count_buf, 3, MPI_INT, rev_id, 0, MPI_COMM_WORLD);
                    // contorl_buf[2] result ready
                    if (control_buf[2] == 1) {
                        //post_work
                        for (int p=0; p<pe_per_node; p++) {
                            int pwid = control_buf[1] + p;
                            if (pwid < max_pe) {
                                for (int pwc=0; pwc<pieces_per_pe; pwc++) {
                                    int poffset = pwid*pieces_per_pe + pwc;
                                    MPI_Recv(node_piece[poffset].voltage, nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    MPI_Recv(node_piece[poffset].charge     , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    /*
                                    for (int i=0; i<wires_per_piece; i++) 
                                        MPI_Recv(wire_piece[poffset].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Recv(wire_piece[poffset].voltages[i], WIRE_SEGMENTS-1, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    MPI_Recv(node_piece[poffset].charge     , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    MPI_Recv(wire_piece[poffset].shr_charge , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                    */
                                }//endfor pwc 
                            }//endif max_pe
                        }//endfor
                    }//endif
                    // identify work portition to new child PE
                    for (int p=0; p<pe_per_node; p++) {
                        wid = count + p;
                        if (wid < max_pe) {
                            for (int c=0; c<pieces_per_pe; c++) {
                                int noffset = wid*pieces_per_pe + c;
                                // send node info
                                MPI_Send(node_piece[noffset].capacitance, nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].leakage    , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].charge     , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].voltage    , nodes_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].shr_pc     , nodes_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(node_piece[noffset].node_attr  , nodes_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].shr_charge , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                // send wire info
                                /*for (int i=0; i<wires_per_piece; i++)
                                    MPI_Send(wire_piece[noffset].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                for (int i=0; i<wires_per_piece; i++)
                                    MPI_Send(wire_piece[noffset].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].resistance , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].inductance , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].capacitance, wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].in_ptr     , wires_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].out_ptr    , wires_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].wire_attr  , wires_per_piece, MPI_INT      , rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].shr_voltage, wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].shr_charge , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                MPI_Send(wire_piece[noffset].shr_pc     , wires_per_piece, MPI_PRECISION, rev_id, 0, MPI_COMM_WORLD);
                                */
                            }//endfor pieces_per_pe
                        }//endif max_pe
                    }//endfor pe_per_node
                    count += pe_per_node;
                    control_buf[0] = 0 ;
                    control_buf[1] = 0;
                    control_buf[2] = 0;
                }//endif work finish
            }//endwhile
            // get final result 
            for (int i=1; i < num_pe; i++) {
                MPI_Recv(control_buf, 4, MPI_INT, i, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                count_buf[1] = 2;
                MPI_Send(count_buf, 3, MPI_INT, i, 0, MPI_COMM_WORLD);
                int inum = i;
                // post_work
                for (int p=0; p<pe_per_node; p++) {
                    int pwid = control_buf[1] + p;
                    if (pwid < max_pe) {
                        for (int pwc=0; pwc<pieces_per_pe; pwc++) {
                            int poffset = pwid*pieces_per_pe + pwc;
                            MPI_Recv(node_piece[poffset].voltage, nodes_per_piece, MPI_PRECISION, inum, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                            MPI_Recv(node_piece[poffset].charge , nodes_per_piece, MPI_PRECISION, inum, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                        }//endfor pwc 
                    }//endif max_pe
                }//endfor

            } 
            // Note: done by single main PE only, post work to update node charge 
            /* Post work by main PE*/
#if 1
            for (int n = 0; n < num_pieces; n++) {
                for (int i=0; i<wires_per_piece; i++) {
                    if (wire_piece[n].wire_attr[i] == 1) {
                        wire_piece[n].shr_voltage[i] =  node_piece[wire_piece[n].shr_pc[i]].voltage[wire_piece[n].out_ptr[i]];
                    }// endif wire_attr
                }// endfor n_num_piece
            } //end for num_pieces
#endif
#ifdef _RESULT
        if (iloop == num_loops -1) {
        for (int n=0; n<num_pieces; n++) {
                for (int it = 0; it<nodes_per_piece; ++it) {
                    printf("\t**node info **\n");
                    printf("\tvoltage: %f, charge: %f\n", node_piece[n].voltage[it], node_piece[n].charge[it]);
                }
                printf("++++++++++++++++++++++++++++++++++++++++++++++++++\n");
        }
        }
#endif

    }//endif rank0
    else {
        int  pieces_per_work = pe_per_node * pieces_per_pe;
            memset(count_buf, 0, sizeof(count_buf));
            memset(child_buf, 0, sizeof(child_buf));
            memset(control_buf, 0, sizeof(control_buf));
            memset(compute_info, 0, sizeof(compute_info));
            end = false;
            control_buf[3] = 1;
            while (!end) {
                // request new child PE
                control_buf[0] = 1;
                MPI_Send(control_buf, 4, MPI_INT, 0, 0, MPI_COMM_WORLD);
                // receive child PE request, wid or complete
                MPI_Recv(count_buf, 3, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                // get wid, begin computation
                if (count_buf[1] == 1) {
                    if (control_buf[2] == 1) {
                        //post_work
                        for (int p=0; p<pe_per_node; p++) {
                            int pwid = wid + p;
                            if (pwid < max_pe) {
                                for (int n = 0; n < pieces_per_pe; n++) {
                                    int npiece = p*pieces_per_pe + n;
                                    MPI_Send(node_piece[npiece].voltage, nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(node_piece[npiece].charge , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    /*
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    */
                                }// end for
                            }//endif
                        }// end for
                    }//endif
                    // init wid
                    wid = count_buf[2];
                    
                    for (int p=0; p<pe_per_node; p++) {
                        int wtid = wid + p;
                        if (wtid < max_pe) {         
                            for (int n = 0; n < pieces_per_pe; n++) {
                                int npiece = p*pieces_per_pe + n;
                                // allocate space for array in soa
                                MPI_Recv(node_piece[npiece].capacitance, nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[npiece].leakage    , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[npiece].voltage    , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[npiece].shr_pc     , nodes_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[npiece].node_attr  , nodes_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                MPI_Recv(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                                /*for (int i=0; i<wires_per_piece; i++)
                                   MPI_Recv(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               for (int i=0; i<wires_per_piece; i++)
                                   MPI_Recv(wire_piece[npiece].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].resistance , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].inductance , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].capacitance, wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].in_ptr     , wires_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].out_ptr    , wires_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].wire_attr  , wires_per_piece, MPI_INT      , 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].shr_voltage, wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               MPI_Recv(wire_piece[npiece].shr_pc     , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                               */
                            }//endfor pieces_per_pe
                        }//endif avail pe
                    }//endfor pe_per_node
                    
                    MPI_Comm   children_comm;
                    // spawn child pe to run
                    snprintf(wid_str, 8, "%d", count_buf[2]);
                    char     * c_argv[] = {const_cast<char *>("workerB"), rank_str, wid_str, NULL};
                    //Each of the processes in the master-job spawns a worker-job
                    //consisting of NUM_WORKER_PROCS processes.
#if 0
                    int offset=0;
                    for (int i=0;i<strlen(hostname);i++) {
                        if (hostname[i] == '.') {
                            offset = i;
                            break;
                        }
                    }
                    hostname[offset] = '\0';
 #endif
                    MPI_Info spawn_info;
                    MPI_Info_create(&spawn_info);
                    MPI_Info_set(spawn_info, "host", hostname);

                    int pe_spn;
                    if (wid+pe_per_node > max_pe)
                        pe_spn = max_pe - wid;
                    else
                        pe_spn = pe_per_node;
                    //DEBUG_R
#ifdef DEBUGER_OUT
                    printf("\t Rank: %d, Creating children working PEs--->\n");
#endif
                    MPI_Comm_spawn(argv[0], c_argv, pe_spn, spawn_info, 0, MPI_COMM_SELF, &children_comm, MPI_ERRCODES_IGNORE);
                    //DEBUG_R
#ifdef DEBUGER_OUT
                    printf("\t Rank: %d, Complete children working PEs--->\n");
#endif
                    compute_info[0] = pieces_per_pe;
                    compute_info[1] = nodes_per_piece;
                    compute_info[2] = wires_per_piece;
                    // send data
                    for (int p=0; p<pe_per_node; p++) {
                        int wtid = wid + p;
                        if (wtid < max_pe) {
                            //send computation info to children PEs
                            MPI_Send(compute_info, 3, MPI_INT, p, 0, children_comm);
                            for (int n = 0; n < pieces_per_pe; n++) {
                                int npiece = p*pieces_per_pe + n;
                                // allocate space for array in soa
                                MPI_Send(node_piece[npiece].capacitance, nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].leakage    , nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].voltage    , nodes_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(node_piece[npiece].shr_pc     , nodes_per_piece, MPI_INT      , p, 0, children_comm);
                                MPI_Send(node_piece[npiece].node_attr  , nodes_per_piece, MPI_INT      , p, 0, children_comm);
                                /*for (int i=0; i<wires_per_piece; i++)
                                   MPI_Send(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, p, 0, children_comm);
                                for (int i=0; i<wires_per_piece; i++)
                                    MPI_Send(wire_piece[npiece].voltages[i], WIRE_SEGMENTS-1, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].resistance , wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].inductance , wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].capacitance, wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].in_ptr     , wires_per_piece, MPI_INT      , p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].out_ptr    , wires_per_piece, MPI_INT      , p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].wire_attr  , wires_per_piece, MPI_INT      , p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].shr_voltage, wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                                MPI_Send(wire_piece[npiece].shr_pc     , wires_per_piece, MPI_PRECISION, p, 0, children_comm);
                                */
                            }//endfor pieces_per_pe
                        }//endif avail pe
                    }//endfor pe_per_node
                    // receive data back
                    for (int p=0; p<pe_per_node; p++) {
                        int wtid = wid + p;
                        if (wtid < max_pe) {
                            // get post_work back
                            for (int n=0; n<pieces_per_pe; n++) {
                                int npiece = p*pieces_per_pe + n;
                                MPI_Recv(node_piece[npiece].voltage, nodes_per_piece, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[npiece].charge, nodes_per_piece, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                /*for (int i=0; i<wires_per_piece; i++)
                                    MPI_Recv(wire_piece[n].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                for (int i=0; i<wires_per_piece; i++)
                                    MPI_Recv(wire_piece[n].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                MPI_Recv(node_piece[n].charge, nodes_per_piece, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                MPI_Recv(wire_piece[n].shr_charge, wires_per_piece, MPI_PRECISION, p, 0, children_comm, MPI_STATUS_IGNORE);
                                */
                            } //end for num_piece
                        } //endif wtid
                    } // end for pe_per_node
                    //Disconnect children PE
                    MPI_Comm_disconnect(&children_comm);
                    // set count_buf[0] to transfer results back to main PE
                    count_buf[0]   = 1; 
                    control_buf[1] = wid;
                    control_buf[2] = 1;
                }//endif computation
                else if (count_buf[1] == 2) {
                    if (control_buf[2] == 1) {
                        for (int p=0; p<pe_per_node; p++) {
                            int wtid = wid + p;
                            if (wtid < max_pe) {
                               for (int n = 0; n < pieces_per_pe; n++) {
                                    int npiece = p*pieces_per_pe + n;
                                    MPI_Send(node_piece[npiece].voltage, nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(node_piece[npiece].charge , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    /*
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    for (int i=0; i<wires_per_piece; i++)
                                        MPI_Send(wire_piece[npiece].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(node_piece[npiece].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    MPI_Send(wire_piece[npiece].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, MPI_COMM_WORLD);
                                    */
                               }// end for
                            }//endif
                        }//endfor 
                    }//endif send final work back
                    control_buf[2] = 0;
                    end = true;
                }//endelseif endflag communication
            }//end while
    }//endelse
#ifdef _RNODE
    printf("End of computing---> PE[%d]\n", rank);
#endif
    //MPI_Finalize();
#endif
}// main forloop
    return 0;
}

int WorkerRoutine(int argc, char ** argv) {
    /* Computing circuit graph */
    MPI_Comm parent_comm;
    int      parent_size;
    int      task_buf[2];
    int      compute_info[3];
    //DEBUG_R
#ifdef DEBUGER_OUT
    printf("\t\tRetrieve parent PEs--->\n");
#endif
    MPI_Comm_get_parent(&parent_comm);
    //DEBUG_R
#ifdef DEBUGER_OUT
    printf("\t\tGet parent PEs--->\n");
#endif
    if (parent_comm == MPI_COMM_NULL) {
        printf("Error: Parent_Proc Unknown!\n");
        return -1;
    }
    //Attention!: The size of the inter-communicator obtained through the
    //            MPI_Comm_remote_size() will always be '1' since a number
    //            of NUM_WORKER_PROCS child processes are spawned by each
    //            of the master processes. Therefore, each group of the
    //            NUM_WORKER_PROCS child processes recognizes only their
    //            correspodning master process in the inter-communicator.
    MPI_Comm_remote_size(parent_comm, &parent_size);
    //output node info
    char hostname[256];
    gethostname(hostname, 256);
#ifdef _RNODE
    printf("\tMy rank: %d, Parent_rank: %d, Node name: %s\n", rank, parent_rank, hostname); 
#endif
    // receive computation info
#if 1
    MPI_Recv(compute_info, 3, MPI_INT, 0, 0, parent_comm, MPI_STATUS_IGNORE);
#else
    MPI_Bcast(compute_info, 2, MPI_INT, MPI_PROC_NULL, parent_comm);
#endif
    int pieces_per_pe = compute_info[0];
    nodes_per_piece   = compute_info[1];
    wires_per_piece   = compute_info[2];
    node * node_piece = new node[pieces_per_pe];
    wire * wire_piece = new wire[pieces_per_pe];
#ifdef DEBUGER_OUT
    printf("cPE  pieces_per_pe: %d, nodes_per_piece: %d\n", pieces_per_pe, nodes_per_piece);
#endif
    // node initialization
    for (int n = 0; n < pieces_per_pe; n++) {
        // allocate space for array in soa
        node_piece[n].capacitance = new PRECISION[nodes_per_piece];
        node_piece[n].leakage     = new PRECISION[nodes_per_piece];
        node_piece[n].charge      = new PRECISION[nodes_per_piece];
        node_piece[n].voltage     = new PRECISION[nodes_per_piece];
        node_piece[n].shr_pc      = new int[nodes_per_piece];
        node_piece[n].node_attr   = new int[nodes_per_piece];
    }
    // wire initialization
    for (int n = 0; n < pieces_per_pe; n++) {
        // allocate space for array in soa of wire
        wire_piece[n].currents    = new PRECISION*[wires_per_piece];
        for (int j=0; j<wires_per_piece; j++)
            wire_piece[n].currents[j]  = new PRECISION[WIRE_SEGMENTS];
        wire_piece[n].voltages    = new PRECISION*[wires_per_piece];
        for (int j=0; j<wires_per_piece; j++)
            wire_piece[n].voltages[j]  = new PRECISION[WIRE_SEGMENTS-1];
        wire_piece[n].resistance  = new PRECISION[wires_per_piece];
        wire_piece[n].inductance  = new PRECISION[wires_per_piece];
        wire_piece[n].capacitance = new PRECISION[wires_per_piece];
        wire_piece[n].in_ptr      = new int[wires_per_piece];
        wire_piece[n].out_ptr     = new int[wires_per_piece];
        wire_piece[n].wire_attr   = new int[wires_per_piece];
        // init wire shared part
        wire_piece[n].shr_voltage = new PRECISION[wires_per_piece];
        wire_piece[n].shr_charge  = new PRECISION[wires_per_piece];
        wire_piece[n].shr_pc      = new int[wires_per_piece];
    }
    /* receive data */
    for (int n = 0; n < pieces_per_pe; n++) {
        // allocate space for array in soa
#if 1
        MPI_Recv(node_piece[n].capacitance, nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].leakage    , nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].voltage    , nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].shr_pc     , nodes_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].node_attr  , nodes_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
#if 1
        for (int i=0; i<wires_per_piece; i++)
            MPI_Recv(wire_piece[n].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        for (int i=0; i<wires_per_piece; i++)
            MPI_Recv(wire_piece[n].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].resistance , wires_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].inductance , wires_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].capacitance, wires_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].in_ptr     , wires_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].out_ptr    , wires_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].wire_attr  , wires_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].shr_voltage, wires_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(wire_piece[n].shr_pc     , wires_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
#endif
#else
#if 0
        MPI_Bcast(node_piece[n].capacitance, nodes_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(node_piece[n].leakage    , nodes_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(node_piece[n].charge     , nodes_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(node_piece[n].voltage    , nodes_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(node_piece[n].shr_pc     , nodes_per_piece, MPI_INT      , MPI_PROC_NULL, parent_comm);
        MPI_Bcast(node_piece[n].node_attr  , nodes_per_piece, MPI_INT      , MPI_PROC_NULL, parent_comm);
        for (int i=0; i<wires_per_piece; i++)
            MPI_Bcast(wire_piece[n].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        for (int i=0; i<wires_per_piece; i++)
            MPI_Bcast(wire_piece[n].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].resistance , wires_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].inductance , wires_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].capacitance, wires_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].in_ptr     , wires_per_piece, MPI_INT      , MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].out_ptr    , wires_per_piece, MPI_INT      , MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].wire_attr  , wires_per_piece, MPI_INT      , MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].shr_voltage, wires_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].shr_charge , wires_per_piece, MPI_PRECISION, MPI_PROC_NULL, parent_comm);
        MPI_Bcast(wire_piece[n].shr_pc     , wires_per_piece, MPI_PRECISION, 0, parent_comm);
#endif
#endif
    }//endfor pieces_per_pe
//DEBUG_R
#if 0
            for (int n=0; n<pieces_per_pe; n++) {
                for (int it=0; it<nodes_per_piece; it++) {
                    printf("\t**node info **\n");
                    printf("\tvoltage: %f, charge: %f\n", node_piece[n].voltage[it], node_piece[n].charge[it]);
                }
            }
#endif

    // GPU: main loop
    // GPU initialization
    PRECISION * d_node_capacitance, * d_node_leakage, * d_node_charge, * d_node_voltage;
    PRECISION * d_wire_currents, * d_wire_voltages, * d_wire_resistance, * d_wire_inductance, * d_wire_capacitance;
    PRECISION * d_shr_voltage, * d_shr_charge;
    int       * d_in_ptr, * d_out_ptr, * d_shr_pc, * d_wire_attr;
    // GPU setDeivce
    // GPU allocation
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_capacitance, sizeof(PRECISION)*nodes_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_leakage    , sizeof(PRECISION)*nodes_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_charge     , sizeof(PRECISION)*nodes_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_voltage    , sizeof(PRECISION)*nodes_per_piece));
        
    cudaCheckError( __LINE__, hipMalloc((void **) &d_wire_currents   , sizeof(PRECISION)*wires_per_piece*WIRE_SEGMENTS));  
    cudaCheckError( __LINE__, hipMalloc((void **) &d_wire_voltages   , sizeof(PRECISION)*wires_per_piece*(WIRE_SEGMENTS-1)));   
    cudaCheckError( __LINE__, hipMalloc((void **) &d_wire_resistance , sizeof(PRECISION)*wires_per_piece));   
    cudaCheckError( __LINE__, hipMalloc((void **) &d_wire_inductance , sizeof(PRECISION)*wires_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_wire_capacitance, sizeof(PRECISION)*wires_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_in_ptr, sizeof(int)*wires_per_piece));    
    cudaCheckError( __LINE__, hipMalloc((void **) &d_out_ptr, sizeof(int)*wires_per_piece));    
    cudaCheckError( __LINE__, hipMalloc((void **) &d_shr_voltage     , sizeof(PRECISION)*wires_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_shr_charge      , sizeof(PRECISION)*wires_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_shr_pc, sizeof(int)*wires_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_wire_attr       , sizeof(int)*wires_per_piece));    
    /* Note: All children-PEs in work-routine are workers, no controller here! */ 
        //if (rank) { 
    /* computation: calculate currents & distributed charge */
    for (int n=0; n<pieces_per_pe; n++) {
        // CPU to GPU memcpy
        cudaCheckError( __LINE__, hipMemcpy( d_node_capacitance, node_piece[n].capacitance, sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_node_leakage    , node_piece[n].leakage    , sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_node_charge     , node_piece[n].charge     , sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_node_voltage    , node_piece[n].voltage    , sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));

        for (int i = 0; i < wires_per_piece; i++) {
            int coffset = i * WIRE_SEGMENTS;
            int voffset = i * (WIRE_SEGMENTS-1);
            cudaCheckError( __LINE__, hipMemcpy( (d_wire_currents+coffset) , wire_piece[n].currents[i] , sizeof(PRECISION)*WIRE_SEGMENTS, hipMemcpyHostToDevice));
            cudaCheckError( __LINE__, hipMemcpy( (d_wire_voltages+voffset) , wire_piece[n].voltages[i] , sizeof(PRECISION)*(WIRE_SEGMENTS-1), hipMemcpyHostToDevice));
        } 
        cudaCheckError( __LINE__, hipMemcpy( d_wire_resistance , wire_piece[n].resistance , sizeof(PRECISION)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_wire_inductance , wire_piece[n].inductance , sizeof(PRECISION)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_wire_capacitance, wire_piece[n].capacitance, sizeof(PRECISION)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_in_ptr          , wire_piece[n].in_ptr     , sizeof(int)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_out_ptr         , wire_piece[n].out_ptr    , sizeof(int)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_shr_voltage     , wire_piece[n].shr_voltage, sizeof(PRECISION)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_shr_charge      , wire_piece[n].shr_charge , sizeof(PRECISION)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_shr_pc          , wire_piece[n].shr_pc     , sizeof(int)*wires_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_wire_attr       , wire_piece[n].wire_attr  , sizeof(int)*wires_per_piece, hipMemcpyHostToDevice));

        // <<<calculate currents>>> gpu
        calculate_current_gpu<<<num_blocks, num_threads>>>(wires_per_piece, d_wire_currents, d_wire_voltages, d_in_ptr, d_out_ptr, d_wire_inductance, d_wire_resistance, d_wire_capacitance, d_node_voltage, d_wire_attr, d_shr_voltage);
        cudaCheckError( __LINE__, hipDeviceSynchronize()); 
        // <<<distributed charge>>> gpu
        distributed_charge_gpu<<<num_blocks, num_threads>>>(wires_per_piece, d_wire_currents,  d_in_ptr, d_out_ptr, d_node_charge, d_wire_attr, d_shr_charge);
        cudaCheckError( __LINE__, hipDeviceSynchronize()); 

        // GPU to CPU memcpy
        cudaCheckError( __LINE__, hipMemcpy( node_piece[n].charge, d_node_charge     ,  sizeof(PRECISION)*nodes_per_piece, hipMemcpyDeviceToHost));
        cudaCheckError( __LINE__, hipMemcpy( wire_piece[n].shr_charge, d_shr_charge      ,  sizeof(PRECISION)*wires_per_piece, hipMemcpyDeviceToHost));
        for (int i = 0; i < wires_per_piece; i++) {
            int coffset = i * WIRE_SEGMENTS;
            int voffset = i * (WIRE_SEGMENTS-1);
            cudaCheckError( __LINE__, hipMemcpy( wire_piece[n].currents[i], (d_wire_currents+coffset) ,  sizeof(PRECISION)*WIRE_SEGMENTS, hipMemcpyDeviceToHost));
            cudaCheckError( __LINE__, hipMemcpy( wire_piece[n].voltages[i], (d_wire_voltages+voffset) ,  sizeof(PRECISION)*(WIRE_SEGMENTS-1), hipMemcpyDeviceToHost));
        }// for wire_per_piece
    } // for: piece_gpu
    /* Note: Post work for charge distribution to send back to parent-PEs to process*/
        // computing PE send post_work to main PE
    for (int n=0; n<pieces_per_pe; n++) {
        for (int i=0; i<wires_per_piece; i++)
            MPI_Send(wire_piece[n].currents[i], WIRE_SEGMENTS  , MPI_PRECISION, 0, 0, parent_comm);
        for (int i=0; i<wires_per_piece; i++)
            MPI_Send(wire_piece[n].voltages[i] , WIRE_SEGMENTS-1, MPI_PRECISION, 0, 0, parent_comm);
        MPI_Send(node_piece[n].charge, nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm);
        MPI_Send(wire_piece[n].shr_charge , wires_per_piece, MPI_PRECISION, 0, 0, parent_comm);
    }
#if 0
            for (int n=0; n<num_pieces; n++) {
                for (int it = 0; it<wires_per_piece; ++it) {
                    printf("\t**node info **\n");
                    printf("\tin_charge: %f, out_charge: %f\n", node_piece[n].charge[wire_piece[n].in_ptr[it]], node_piece[n].charge[wire_piece[n].out_ptr[it]]);
                }
            }
            printf("++++++++++++++++++++++++++++++++++++++++++++++++++\n");
#endif

    /* free cudamem */
    // GPU deallocation
    cudaCheckError( __LINE__, hipFree(d_node_capacitance));    
    cudaCheckError( __LINE__, hipFree(d_node_leakage));    
    cudaCheckError( __LINE__, hipFree(d_node_charge));    
    cudaCheckError( __LINE__, hipFree(d_node_voltage));    
    cudaCheckError( __LINE__, hipFree(d_wire_currents)); 
    cudaCheckError( __LINE__, hipFree(d_wire_voltages));   
    cudaCheckError( __LINE__, hipFree(d_wire_resistance));   
    cudaCheckError( __LINE__, hipFree(d_wire_inductance));
    cudaCheckError( __LINE__, hipFree(d_wire_capacitance));    
    cudaCheckError( __LINE__, hipFree(d_in_ptr));    
    cudaCheckError( __LINE__, hipFree(d_out_ptr));    
    cudaCheckError( __LINE__, hipFree(d_shr_voltage));    
    cudaCheckError( __LINE__, hipFree(d_shr_charge));    
    cudaCheckError( __LINE__, hipFree(d_shr_pc));    
    cudaCheckError( __LINE__, hipFree(d_wire_attr));    
    // MPI_Comm release
#ifdef DEBUGER_OUT
    printf("\tfree parent_comm---> children PE[%d]\n", rank);
#endif
    //MPI_Comm_free(&parent_comm);
#ifdef _RNODE
    printf("\tEnd of computing---> children PE[%d]\n", rank);
#endif
    //Disconnect children PE
    MPI_Comm_disconnect(&parent_comm);
    //MPI_Finalize();

    return 0;
}

int WorkerRoutine_B(int argc, char ** argv) {
    /* Computing circuit graph */
    MPI_Comm parent_comm;
    int      parent_size;
    int      task_buf[2];
    int      compute_info[3];

    //DEBUG_R
#ifdef DEBUGER_OUT
    printf("\t\tRetrieve parent PEs--->\n");
#endif
    MPI_Comm_get_parent(&parent_comm);
    //DEBUG_R
#ifdef DEBUGER_OUT
    printf("\t\tGet parent PEs--->\n");
#endif
    if (parent_comm == MPI_COMM_NULL)
        return -1;
    //Attention!: The size of the inter-communicator obtained through the
    //            MPI_Comm_remote_size() will always be '1' since a number
    //            of NUM_WORKER_PROCS child processes are spawned by each
    //            of the master processes. Therefore, each group of the
    //            NUM_WORKER_PROCS child processes recognizes only their
    //            correspodning master process in the inter-communicator.
    MPI_Comm_remote_size(parent_comm, &parent_size);

    //output node info
    char hostname[256];
    gethostname(hostname, 256);
#ifdef _RNODE
    printf("\tMy rank: %d, Parent_rank: %d, Node name: %s\n", rank, parent_rank, hostname); 
#endif
    // receive computation info
    MPI_Recv(compute_info, 3, MPI_INT, 0, 0, parent_comm, MPI_STATUS_IGNORE);
    int pieces_per_pe = compute_info[0];
    nodes_per_piece   = compute_info[1];
    wires_per_piece   = compute_info[2];
    node * node_piece = new node[pieces_per_pe];
    wire * wire_piece = new wire[pieces_per_pe];

    // node initialization
    for (int n = 0; n < pieces_per_pe; n++) {
        // allocate space for array in soa
        node_piece[n].capacitance = new PRECISION[nodes_per_piece];
        node_piece[n].leakage     = new PRECISION[nodes_per_piece];
        node_piece[n].charge      = new PRECISION[nodes_per_piece];
        node_piece[n].voltage     = new PRECISION[nodes_per_piece];
        node_piece[n].shr_pc      = new int[nodes_per_piece];
        node_piece[n].node_attr   = new int[nodes_per_piece];
    }
    /* receive data */
    for (int n = 0; n < pieces_per_pe; n++) {
        // allocate space for array in soa
        MPI_Recv(node_piece[n].capacitance, nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].leakage    , nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].charge     , nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].voltage    , nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].shr_pc     , nodes_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
        MPI_Recv(node_piece[n].node_attr  , nodes_per_piece, MPI_INT      , 0, 0, parent_comm, MPI_STATUS_IGNORE);
    }//endfor pieces_per_pe

    // GPU: main loop
    // GPU initialization
    PRECISION * d_node_capacitance, * d_node_leakage, * d_node_charge, * d_node_voltage;
    //PRECISION * d_wire_currents, * d_wire_voltages, * d_wire_resistance, * d_wire_inductance, * d_wire_capacitance;
    PRECISION * d_shr_voltage, * d_shr_charge;
    int       * d_in_ptr, * d_out_ptr, * d_shr_pc, * d_wire_attr;
    // GPU setDeivce
    // GPU allocation
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_capacitance, sizeof(PRECISION)*nodes_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_leakage    , sizeof(PRECISION)*nodes_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_charge     , sizeof(PRECISION)*nodes_per_piece));
    cudaCheckError( __LINE__, hipMalloc((void **) &d_node_voltage    , sizeof(PRECISION)*nodes_per_piece));
        
    /* Note: GPU main loop needed to be moved to main PE*/
    /* Note: All children-PEs in work-routine are workers, no controller here! */ 
    /* computation: update voltage */
    for (int n=0; n<pieces_per_pe; n++) {
        // CPU to GPU memcpy
        cudaCheckError( __LINE__, hipMemcpy( d_node_capacitance, node_piece[n].capacitance, sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_node_leakage    , node_piece[n].leakage    , sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_node_charge     , node_piece[n].charge     , sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));
        cudaCheckError( __LINE__, hipMemcpy( d_node_voltage    , node_piece[n].voltage    , sizeof(PRECISION)*nodes_per_piece, hipMemcpyHostToDevice));
        // update voltage gpu
        update_voltage_gpu<<<num_blocks, num_threads>>>(nodes_per_piece, d_node_voltage, d_node_charge, d_node_capacitance, d_node_leakage);
        cudaCheckError( __LINE__, hipDeviceSynchronize()); 
        // GPU to CPU memcpy
        cudaCheckError( __LINE__, hipMemcpy( node_piece[n].charge, d_node_charge, sizeof(PRECISION)*nodes_per_piece, hipMemcpyDeviceToHost));
        cudaCheckError( __LINE__, hipMemcpy( node_piece[n].voltage, d_node_voltage, sizeof(PRECISION)*nodes_per_piece, hipMemcpyDeviceToHost));
    } // for: piece_gpu
    /* Note: Post work for charge distribution to send back to parent-PEs to process*/
        // computing PE send post_work to main PE
    for (int n=0; n<pieces_per_pe; n++) {
        MPI_Send(node_piece[n].voltage, nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm);
        MPI_Send(node_piece[n].charge , nodes_per_piece, MPI_PRECISION, 0, 0, parent_comm);
    }
    /* free cudamem */
    // GPU deallocation
    cudaCheckError( __LINE__, hipFree(d_node_capacitance));    
    cudaCheckError( __LINE__, hipFree(d_node_leakage));    
    cudaCheckError( __LINE__, hipFree(d_node_charge));    
    cudaCheckError( __LINE__, hipFree(d_node_voltage));    
    // MPI_Comm release
#ifdef DEBUGER_OUT
    printf("\tfree parent_comm---> second-phase children PE[%d]\n", rank);
#endif
    //MPI_Comm_free(&parent_comm);
#ifdef _RNODE
    printf("\tEnd of computing---> second-phase children PE[%d]\n", rank);
#endif
    //Disconnect children PE
    MPI_Comm_disconnect(&parent_comm);
    //MPI_Finalize();
    return 0;
}


























































